#include "hip/hip_runtime.h"
#include "gabor.cuh"

using namespace std;

namespace CuGabor {
    __global__ void multiply(CuComplex* A, int row_size, int col_size, CuComplex* B, CuComplex* result, int batch_size) {
        int i = threadIdx.x + blockIdx.x * blockDim.x;
        int j = threadIdx.y + blockIdx.y * blockDim.y;

        int index = i*col_size*batch_size+j;
        double a,b,c,d;
        if (i < row_size && j < batch_size*col_size) {
            a = A[index].x;
            b = A[index].y;
            c = B[index].x;
            d = B[index].y;

            result[index].x = a*c - b*d;
            result[index].y = a*d + b*c;
        }
    }

    void cmat_mult(CuComplex* A, int row_size, int col_size, CuComplex* B, CuComplex* result, int batch_size){
        int block = 16;
        dim3 threadsPerBlock(block, block);
        dim3 blocksPerGrid(ceil(row_size/block)+1, ceil(col_size*batch_size/block)+1);

        multiply<<<blocksPerGrid, threadsPerBlock>>>(A, row_size, col_size, B, result, batch_size);

        // Wait for device to finish all operation
        hipDeviceSynchronize();

        // Check if kernel execution generated and error
        //getLastCudaError("Kernel execution failed [ solvePoisson ]");
        hipError_t err = hipGetLastError();   
        if ( err != hipSuccess ){
                //fprintf(stderr, "Kernel execution failed [ solvePoisson ]\n");
                printf("CUDA Error: %s\n", hipGetErrorString(err));   
                return;	
        }
    }

    __global__ void multiply(hipfftDoubleComplex* A, int row_size, int col_size, hipfftDoubleComplex* B, hipfftDoubleComplex* result, int batch_size) {
        int i = threadIdx.x + blockIdx.x * blockDim.x;
        int j = threadIdx.y + blockIdx.y * blockDim.y;

        int index = i*col_size*batch_size+j;
        double a,b,c,d;
        if (i < row_size && j < batch_size*col_size) {
            a = A[index].x;
            b = A[index].y;
            c = B[index].x;
            d = B[index].y;

            result[index].x = a*c - b*d;
            result[index].y = a*d + b*c;
        }
    }

    void cmat_mult(hipfftDoubleComplex* A, int row_size, int col_size, hipfftDoubleComplex* B, hipfftDoubleComplex* result, int batch_size){
        int block = 16;
        dim3 threadsPerBlock(block, block);
        dim3 blocksPerGrid(ceil(row_size/block)+1, ceil(col_size*batch_size/block)+1);

        multiply<<<blocksPerGrid, threadsPerBlock>>>(A, row_size, col_size, B, result, batch_size);

        // Wait for device to finish all operation
        hipDeviceSynchronize();

        // Check if kernel execution generated and error
        //getLastCudaError("Kernel execution failed [ solvePoisson ]");
        hipError_t err = hipGetLastError();   
        if ( err != hipSuccess ){
                //fprintf(stderr, "Kernel execution failed [ solvePoisson ]\n");
                printf("CUDA Error: %s\n", hipGetErrorString(err));   
                return;	
        }
    }

    void conv_dud_gpu_fft(double* out, 
                            const unsigned int* image, 
                            double* kernel, 
                            int image_n, int image_m, int kernel_n, int kernel_m){

        
        
        int batch_size = 1;

        // calculate new size of image based on padding size
        int row_size = image_m + kernel_m - 1;
        int col_size = image_n + kernel_n - 1;
        int size = row_size * col_size;

        if ((2 * size * batch_size) >= CUFFT_MAX_SIZE) {
            throw invalid_argument("Batch of images is too large. The maximumum number of values in cuFFT is 2^27.");
        }

        // allocate space for linear indexed arrays
        Complex* linear_image = (Complex*)malloc(size * batch_size * sizeof(Complex));
        Complex* result = (Complex*)malloc(size * batch_size * sizeof(Complex));
        Complex* linear_kernel = (Complex*)malloc(size * batch_size * sizeof(Complex));

        int index, index2;
        
        for (int i = 0; i < row_size; ++i) {
            for (int j = 0; j < col_size; ++j) {
                index = (i*col_size + j);
                linear_image[index].y = 0.f;
                if (i < image_m && j < image_n) { 
                    index2 = (i*image_n + j) ;
                    linear_image[index].x = image[index2];
                } else {
                    linear_image[index].x = 0.f; // add padding
                }
            }
        }

        for (int i = 0; i < row_size; ++i) {
            for (int j = 0; j < col_size; ++j) {
                index = (i*col_size + j);
                index2 = (i*kernel_n + j);
                if (i < kernel_m && j < kernel_n) {
                    linear_kernel[index].x = kernel[2*index2];
                    //linear_kernel[index].y = kernel[2*index2+1];
                } else {
                    linear_kernel[index].x = 0.f;
                    linear_kernel[index].y = 0.f;
                }
            }
        }

        for (int i = 0; i < row_size; ++i) {
            for (int j = 0; j < col_size; ++j) {
                index = (i*col_size + j);
            }
        }


        CuComplex* d_image;
        CuComplex* d_result;
        CuComplex* d_kernel;

        int n[2] = {row_size, col_size};

        hipMalloc((void**)&d_image, sizeof(CuComplex)*size*batch_size);
        if (hipGetLastError() != hipSuccess){
            fprintf(stderr, "Cuda error: Failed to allocate\n");	
        }

        hipMalloc((void**)&d_result, sizeof(CuComplex)*size*batch_size);
        if (hipGetLastError() != hipSuccess){
            fprintf(stderr, "Cuda error: Failed to allocate\n");	
        }
        
        hipMalloc((void**)&d_kernel, sizeof(CuComplex)*size*batch_size);
        if (hipGetLastError() != hipSuccess){
            fprintf(stderr, "Cuda error: Failed to allocate\n");	
        }
        
        // copy data to GPU
        hipMemcpy(d_image, linear_image, batch_size*size*sizeof(CuComplex), hipMemcpyHostToDevice);
        if (hipGetLastError() != hipSuccess){
            fprintf(stderr, "Cuda error: Failed to allocate\n");	
        }
        hipMemcpy(d_kernel, linear_kernel, batch_size*size*sizeof(CuComplex), hipMemcpyHostToDevice);
        if (hipGetLastError() != hipSuccess){
            fprintf(stderr, "Cuda error: Failed to allocate\n");	
        }

        hipfftHandle plan;
        hipfftHandle plan_k;
        int idist = size;
        int odist = size;
        
        int inembed[] = {row_size, col_size};
        int onembed[] = {row_size, col_size};

        int istride = 1;
        int ostride = 1;

        if (hipfftPlanMany(&plan, 2, n, inembed, istride, idist, onembed, ostride, odist, HIPFFT_C2C, batch_size) != HIPFFT_SUCCESS){
            fprintf(stderr, "CUFFT Error: Unable to create plan\n");
            //return;	
        }
        if (hipfftPlanMany(&plan_k, 2, n, inembed, istride, idist, onembed, ostride, odist, HIPFFT_C2C, batch_size) != HIPFFT_SUCCESS){
            fprintf(stderr, "CUFFT Error: Unable to create plan\n");
            //return;	
        }

        if (hipfftExecC2C(plan, d_image, d_image, HIPFFT_FORWARD) != HIPFFT_SUCCESS){
            fprintf(stderr, "CUFFT Error: Unable to execute plan\n");
            //return;		
        }

        if (hipfftExecC2C(plan_k, d_kernel, d_kernel, HIPFFT_FORWARD) != HIPFFT_SUCCESS){
            fprintf(stderr, "CUFFT Error: Unable to execute plan\n");
            //return;		
        }

        if (hipDeviceSynchronize() != hipSuccess){
            fprintf(stderr, "Cuda error: Failed to synchronize\n");
            //return;
        }

        // element-wise multiplication of the image and kernel
        cmat_mult(d_image, row_size, col_size, d_kernel, d_result, batch_size);

        // transform out of fourier space
        if (hipfftExecC2C(plan, d_result, d_result, HIPFFT_BACKWARD) != HIPFFT_SUCCESS){
            fprintf(stderr, "CUFFT Error: Unable to execute plan\n");
            //return;		
        }

        // copy results from device to host
        hipMemcpy(result, d_result, batch_size*size*sizeof(CuComplex), hipMemcpyDeviceToHost); 

        // transfer to output array 
        for(int i = 0; i < size; ++i) {
            out[2*i] = (result[i].x/(size));
            out[2*i + 1] = (result[i].y/(size));
        }
        
        // free host memory
        free(linear_image);
        free(result);
        free(linear_kernel);

        // free device memory
        hipfftDestroy(plan);
        hipfftDestroy(plan_k);
        hipFree(d_image);
        hipFree(d_result);
        hipFree(d_kernel);
                            
    }

     void conv_dud_gpu_fft_multi_filter(double* out, 
                            const unsigned int* image, 
                            double* kernel, 
                            int image_n, int image_m, int kernel_n, int kernel_m, int batch_size){
        
        // calculate new size of image based on padding size
        int row_size = image_m + kernel_m - 1;
        int col_size = image_n + kernel_n - 1;
        int size = row_size * col_size;

        if ((2 * size * batch_size) >= CUFFT_MAX_SIZE) {
            throw invalid_argument("Batch of images is too large. The maximumum number of values in cuFFT is 2^27.");
        }

        // allocate space for linear indexed arrays
        Complex* linear_image = (Complex*)malloc(size * batch_size * sizeof(Complex));
        Complex* result = (Complex*)malloc(size * batch_size * sizeof(Complex));
        Complex* linear_kernel = (Complex*)malloc(size * batch_size * sizeof(Complex));

        int index, index2;
        
        int batch_idx, batch_idx2;
        for (int batch = 0; batch < batch_size; ++batch) {
            batch_idx = batch * size;
            //batch_idx2 = batch * image_m * image_n;
            for (int i = 0; i < row_size; ++i) {
                for (int j = 0; j < col_size; ++j) {
                    index = batch_idx + (i*col_size + j);
                    linear_image[index].y = 0.f;
                    if (i < image_m && j < image_n) { 
                        //index2 = batch_idx2 + (i*image_n + j) ;
                        index2 = (i*image_n + j);
                        linear_image[index].x = image[index2];
                    } else {
                        linear_image[index].x = 0; // add padding
                    }
                }
            }
        }

        for (int batch = 0; batch < batch_size; ++batch) {
            batch_idx = batch * size;
            batch_idx2 = batch * kernel_m * kernel_n;
            for (int i = 0; i < row_size; ++i) {
                for (int j = 0; j < col_size; ++j) {
                    index = batch_idx + (i*col_size + j);
                    index2 = batch_idx2 + (i*kernel_n + j);
                    if (i < kernel_m && j < kernel_n) {
                        linear_kernel[index].x = kernel[2*index2];
                        linear_kernel[index].y = kernel[2*index2+1];
                    } else {
                        linear_kernel[index].x = 0;
                        linear_kernel[index].y = 0;
                    }
                }
            }
        }

        CuComplex* d_image;
        CuComplex* d_result;
        CuComplex* d_kernel;

        int n[2] = {row_size, col_size};

        hipMalloc((void**)&d_image, sizeof(CuComplex)*size*batch_size);
        if (hipGetLastError() != hipSuccess){
            fprintf(stderr, "Cuda error: Failed to allocate\n");	
        }

        hipMalloc((void**)&d_result, sizeof(CuComplex)*size*batch_size);
        if (hipGetLastError() != hipSuccess){
            fprintf(stderr, "Cuda error: Failed to allocate\n");	
        }
        
        hipMalloc((void**)&d_kernel, sizeof(CuComplex)*size*batch_size);
        if (hipGetLastError() != hipSuccess){
            fprintf(stderr, "Cuda error: Failed to allocate\n");	
        }
        
        // copy data to GPU
        hipMemcpy(d_image, linear_image, batch_size*size*sizeof(CuComplex), hipMemcpyHostToDevice);
        if (hipGetLastError() != hipSuccess){
            fprintf(stderr, "Cuda error: Failed to allocate\n");	
        }
        hipMemcpy(d_kernel, linear_kernel, batch_size*size*sizeof(CuComplex), hipMemcpyHostToDevice);
        if (hipGetLastError() != hipSuccess){
            fprintf(stderr, "Cuda error: Failed to allocate\n");	
        }

        hipfftHandle plan;
        hipfftHandle plan_k;
        int idist = size;
        int odist = size;
        
        int inembed[] = {row_size, col_size};
        int onembed[] = {row_size, col_size};

        int istride = 1;
        int ostride = 1;

        if (hipfftPlanMany(&plan, 2, n, inembed, istride, idist, onembed, ostride, odist, HIPFFT_C2C, batch_size) != HIPFFT_SUCCESS){
            fprintf(stderr, "CUFFT Error: Unable to create plan\n");
            //return;	
        }
        if (hipfftPlanMany(&plan_k, 2, n, inembed, istride, idist, onembed, ostride, odist, HIPFFT_C2C, batch_size) != HIPFFT_SUCCESS){
            fprintf(stderr, "CUFFT Error: Unable to create plan\n");
            //return;	
        }

        if (hipfftExecC2C(plan, d_image, d_image, HIPFFT_FORWARD) != HIPFFT_SUCCESS){
            fprintf(stderr, "CUFFT Error: Unable to execute plan\n");
            //return;		
        }

        if (hipfftExecC2C(plan_k, d_kernel, d_kernel, HIPFFT_FORWARD) != HIPFFT_SUCCESS){
            fprintf(stderr, "CUFFT Error: Unable to execute plan\n");
            //return;		
        }

        if (hipDeviceSynchronize() != hipSuccess){
            fprintf(stderr, "Cuda error: Failed to synchronize\n");
            //return;
        }

        // element-wise multiplication of the image and kernel
        cmat_mult(d_image, row_size, col_size, d_kernel, d_result, batch_size);

        // transform out of fourier space
        if (hipfftExecC2C(plan, d_result, d_result, HIPFFT_BACKWARD) != HIPFFT_SUCCESS){
            fprintf(stderr, "CUFFT Error: Unable to execute plan\n");
            //return;		
        }

        // copy results from device to host
        hipMemcpy(result, d_result, batch_size*size*sizeof(CuComplex), hipMemcpyDeviceToHost); 

        
        // transfer to output array 
        for(int batch = 0; batch < batch_size; ++batch){
            batch_idx = batch*size;
            for(int i = 0; i < size; ++i) {
                out[2*batch_idx + 2*i] = (result[batch_idx + i].x/((double)size));
                out[2*batch_idx + 2*i + 1] = (result[batch_idx + i].y/((double)size));
            }
        }
        
        
        // free host memory
        free(linear_image);
        free(result);
        free(linear_kernel);

        // free device memory
        hipfftDestroy(plan);
        hipfftDestroy(plan_k);
        hipFree(d_image);
        hipFree(d_result);
        hipFree(d_kernel);
                            
    }


    __global__ void conv_dud_gpu_helper(double* result, const unsigned int* image, CuComplex* kernel, int col_size, int row_size, int kernel_n, int kernel_m, int kernel_offset){

        // calculate row and column positions
        int row = blockIdx.x * blockDim.x + threadIdx.x;
        int col = blockIdx.y * blockDim.y + threadIdx.y;

        // check bounds
        if(row >= row_size || col >= col_size) return;

        int iFlip, jFlip; // flipped kernel indices
        int ii, jj;
        float temp_x = 0, temp_y=0;

        for(int i = 0; i < kernel_m; ++i){

            iFlip = kernel_m - 1 - i;

            for(int j = 0; j < kernel_n; ++j){

                jFlip = kernel_n - 1 - j;

                ii = row + (kernel_offset - iFlip);
                jj = col + (kernel_offset - jFlip);

                if(ii >= 0 && ii < row_size && jj >= 0 && jj < col_size) {
                    temp_x += image[ii * col_size + jj] * kernel[iFlip * kernel_n + jFlip].x;
                    temp_y += image[ii * col_size + jj] * kernel[iFlip * kernel_n + jFlip].y;
                }
            }
        }

        result[2*(row * col_size + col)] = temp_x;
        result[2*(row * col_size + col)+1] = temp_y;
    }

    void conv_dud_gpu(double* out, 
            const unsigned int* image, 
            double* kernel, 
            int image_n, int image_m, int kernel_n, int kernel_m){
        
        int batch_size = 1;

        hipError_t err;

        // calculate new size of image based on padding size
        int row_size = image_m + kernel_m - 1;
        int col_size = image_n + kernel_n - 1;
        int size = row_size * col_size;
        
        // allocate space for linear indexed arrays
        unsigned int* linear_image = (unsigned int*)malloc(size * sizeof(unsigned int));
        Complex* linear_kernel = (Complex*)malloc(kernel_n*kernel_m * sizeof(Complex));
        
        int index, index2;
        for (int i = 0; i < row_size; ++i) {
            for (int j = 0; j < col_size; ++j) {
                index = (i*col_size + j);
                if (i < image_m && j < image_n) { 
                    index2 = (i*image_n + j) ;
                    linear_image[index] = image[index2];
                } else {
                    linear_image[index] = 0.f; // add padding
                }
            }
        }
        
        for (int i = 0; i < kernel_m; ++i) {
            for (int j = 0; j < kernel_n; ++j) {
                index = (i*kernel_n + j);
                
                linear_kernel[index].x = kernel[2*index];
                linear_kernel[index].y = kernel[2*index+1];
            }
        }

        double* d_result;
        CuComplex* d_kernel;
        unsigned int* d_image;

        // allocate vectors for GPU
        hipMalloc(&d_image, 2*size*sizeof(unsigned int));
        err = hipGetLastError();
        if ( err != hipSuccess )
        {
            printf("CUDA Error: %s\n", hipGetErrorString(err));       
        }

        hipMalloc(&d_result, 2*size*sizeof(double));
        err = hipGetLastError();
        if ( err != hipSuccess )
        {
            printf("CUDA Error: %s\n", hipGetErrorString(err));       
        }

        hipMalloc(&d_kernel, kernel_n*kernel_m*sizeof(CuComplex));
        err = hipGetLastError();
        if ( err != hipSuccess )
        {
            printf("CUDA Error: %s\n", hipGetErrorString(err));       
        }
        
        // copy data to GPU
        hipMemcpy(d_image, linear_image, 2*size*sizeof(unsigned int), hipMemcpyHostToDevice);
        err = hipGetLastError();
        if ( err != hipSuccess )
        {
            printf("CUDA Error: %s\n", hipGetErrorString(err));       
        }

        hipMemcpy(d_kernel, linear_kernel, kernel_n*kernel_m*sizeof(CuComplex), hipMemcpyHostToDevice);
        err = hipGetLastError();
        if ( err != hipSuccess )
        {
            printf("CUDA Error: %s\n", hipGetErrorString(err));       
        }

        //cout << "threads: " << threads << endl;
        int X_THREADS = 16;
        int Y_THREADS = X_THREADS;
        int X_BLOCKS = (col_size + X_THREADS - 1) / X_THREADS;
        int Y_BLOCKS = (row_size + Y_THREADS - 1) / Y_THREADS;

        //cout << X_BLOCKS << endl;
        //cout << Y_BLOCKS << endl;

        string str;

        //cin >> str;

        dim3 block_dim(X_THREADS, Y_THREADS);
        dim3 grid_dim(X_BLOCKS, Y_BLOCKS);

        int offset = kernel_n / 2.; // center of filter
        
        // call kernel
        //conv_dud_gpu_helper(double* result, const unsigned int* image, Complex* kernel, int col_size, int row_size, int kernel_n, int kernel_m, int kernel_offset){
        conv_dud_gpu_helper<<<grid_dim, block_dim>>>(d_result, d_image, d_kernel, col_size, row_size, kernel_n, kernel_m, offset);

        hipDeviceSynchronize();

        err = hipGetLastError();
        if ( err != hipSuccess )
        {
            printf("CUDA Error at sync: %s\n", hipGetErrorString(err));       
        }

        // copy results from device to host
        hipMemcpy(out, d_result, 2*size*sizeof(double), hipMemcpyDeviceToHost);
        err = hipGetLastError();
        if ( err != hipSuccess )
        {
            printf("CUDA Error at memcpy: %s\n", hipGetErrorString(err));       
        }

        // free host memory
        free(linear_image);
        
        free(linear_kernel);

        // free device memory
        hipFree(d_image);
        err = hipGetLastError();
        if ( err != hipSuccess )
        {
            printf("CUDA Error: %s\n", hipGetErrorString(err));       
        }

        hipFree(d_result);
        err = hipGetLastError();
        if ( err != hipSuccess )
        {
            printf("CUDA Error: %s\n", hipGetErrorString(err));       
        }

        hipFree(d_kernel);
        err = hipGetLastError();
        if ( err != hipSuccess )
        {
            printf("CUDA Error: %s\n", hipGetErrorString(err));       
        }
        
    }
}


